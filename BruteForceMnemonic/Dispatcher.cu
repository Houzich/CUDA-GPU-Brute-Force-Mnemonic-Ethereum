#include "hip/hip_runtime.h"
﻿/**
  ******************************************************************************
  * @author		Anton Houzich
  * @version	V1.0.0
  * @date		2-April-2023
  * @mail		houzich_anton@mail.ru
  * discussion  https://t.me/BRUTE_FORCE_CRYPTO_WALLET
  ******************************************************************************
  */


#include <stdafx.h>

#include <iostream>
#include <chrono>
#include <thread>
#include <fstream>
#include <string>
#include <memory>
#include <sstream>
#include <iomanip>
#include <vector>
#include <map>
#include <omp.h>



#include "Dispatcher.h"
#include "GPU.h"
#include "KernelStride.hpp"
#include "Helper.h"


#include "hip/hip_runtime.h"
#include ""


#include "../Tools/tools.h"
#include "../Tools/utils.h"
#include "../config/Config.hpp"


class SaveClass {
	uint32_t _num_file;

public:
	DataClass *dt;
	ConfigClass *cfg;
	size_t count_line;
	std::string file_path;
public:
	SaveClass(DataClass *data, ConfigClass *config)
	{
		dt = data;
		cfg = config;
		count_line = 0;
		setNumFile(0);
	}

public:
	void setFilePath(uint32_t num)
	{
		std::ostringstream ostr;
		ostr << std::setfill('0') << std::setw(4) << num;
		file_path = cfg->folder_save_result + "\\" + ostr.str() + ".csv";
	}
	void setNumFile(uint32_t num)
	{
		_num_file = num;
		setFilePath(num);
	}
	uint32_t getNumFile() {
		return _num_file;
	}
	void incNumFile()
	{
		_num_file++;
		setFilePath(_num_file);
	}
};

void clearFiles(SaveClass* sv) {
	std::ofstream out;
	uint32_t num = sv->getNumFile();
	for (int i = 0; i < 100; i++)
	{
		sv->setNumFile(i);
		out.open(sv->file_path);
		out.close();
	}
	sv->setNumFile(num);
}



void saveResult(SaveClass* sv) {
	std::ofstream out;
	if (sv->count_line + sv->dt->wallets_in_round_gpu > sv->cfg->lines_in_file_save) {
		size_t cnt_line = 0;
		size_t remaining_lines = sv->dt->wallets_in_round_gpu;
		size_t lines = sv->cfg->lines_in_file_save - sv->count_line;
		while (remaining_lines != 0) {
			out.open(sv->file_path, std::ios::app);
			out.write((char*)sv->dt->host.save + cnt_line * SIZE_SAVE_FRAME, lines * SIZE_SAVE_FRAME);
			out.close();
			cnt_line += lines;
			remaining_lines -= lines;
			if(remaining_lines) sv->incNumFile();
			sv->count_line = lines;
			if (remaining_lines > sv->cfg->lines_in_file_save) lines = sv->cfg->lines_in_file_save;
			else lines = remaining_lines;
		}
	}
	else
	{
		out.open(sv->file_path, std::ios::app);
		out.write((char*)sv->dt->host.save, sv->dt->size_save_buf);
		out.close();
		sv->count_line += sv->dt->wallets_in_round_gpu;
	}
	if (sv->count_line >= sv->cfg->lines_in_file_save)
	{
		sv->count_line = sv->count_line - sv->cfg->lines_in_file_save;
		sv->incNumFile();
	}
}

static std::thread save_thread;

int Generate_Mnemonic(void)
{
	hipError_t cudaStatus = hipSuccess;
	ConfigClass Config;
	try {
		parse_config(&Config, "config.cfg");
	}
	catch (...) {
		for (;;)
			std::this_thread::sleep_for(std::chrono::seconds(30));
	}

	devicesInfo();
	// Choose which GPU to run on, change this on a multi-GPU system.
	uint32_t num_device = 0;
#ifndef TEST_MODE
	std::cout << "\n\nEnter number of device: ";
	std::cin >> num_device;
#endif //GENERATE_INFINITY
	cudaStatus = hipSetDevice(num_device);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		return -1;
	}

	size_t num_wallets_gpu = Config.cuda_grid * Config.cuda_block;
	//18,446,744,073,709,551,615
	size_t number_of_addresses = 0;
	size_t count_save_data_in_file = 0;
	int num_bytes = 0;

	std::cout << "\nNUM WALLETS IN ROUND GPU: " << tools::formatWithCommas(num_wallets_gpu) << std::endl << std::endl;
#ifndef TEST_MODE
	std::cout << "Max value: 18,000,000,000,000,000,000 (18000000000000000000)" << std::endl;
	std::cout << "Enter number of generate mnemonic: ";
	std::cin >> number_of_addresses;
	std::cin.ignore(std::numeric_limits<std::streamsize>::max(), '\n');
	number_of_addresses = (((number_of_addresses - 1) / (num_wallets_gpu)+1) * (num_wallets_gpu));

	std::string answer = "";
	while ((answer != "Y") && (answer != "y") && (answer != "N") && (answer != "n"))
	{
		answer = "";
		std::cout << "Save data in files? [Y/n] : ";
		std::getline(std::cin, answer);
		//std::cin.ignore(std::numeric_limits<std::streamsize>::max(), '\n');
	}
	if (answer == "Y" || answer == "y")
	{
		count_save_data_in_file = 0xFFFFFFFFFFFFFFFF;
	}


	std::cout << "Enter num bytes for check 6...8: ";
	std::cin >> num_bytes;
	if (num_bytes != 0)
		if ((num_bytes < 6) || (num_bytes > 8)) {
			std::cout << "Error num bytes. Won't be used!" << std::endl;
			num_bytes = 0;
		}


#else
	number_of_addresses = num_wallets_gpu*15;
	num_bytes = 6;
	count_save_data_in_file = 15;
#endif //TEST_MODE

	DataClass* Data = new DataClass();
	KernelStrideClass* Stride = new KernelStrideClass(Data);
	SaveClass* Save = new SaveClass(Data, &Config);
	std::cout << "READ TABLES! WAIT..." << std::endl;
	clearFiles(Save);
	int err = tools::readAllTables(Data->host.tables, Config.folder_database, "");
	if (err == -1) {
		std::cout << "Error readAllTables!" << std::endl;
		goto Error;
	}


	if (Data->malloc(Config.cuda_grid, Config.cuda_block, count_save_data_in_file == 0 ? false : true) != 0) {
		std::cout << "Error Data->Malloc()!" << std::endl;
		goto Error;
	}

	if (Stride->init() != 0) {
		printf("Error INIT!!\n");
		goto Error;
	}

	Data->host.freeTableBuffers();

	std::cout << "START GENERATE ADDRESSES!" << std::endl;
	std::cout << "PATH: m/44'/60'/0'/0/0.." << (NUM_CHILDS - 1) << ", m/44'/60'/0'/1/0.." << (NUM_CHILDS - 1) << std::endl;
	std::cout << "\nGENERATE " << tools::formatWithCommas(number_of_addresses) << " MNEMONICS. " << tools::formatWithCommas(number_of_addresses * NUM_ALL_CHILDS) << " ADDRESSES. MNEMONICS IN ROUNDS " << tools::formatWithCommas(Data->wallets_in_round_gpu) << ". WAIT...\n\n";

	tools::generateRandomUint64Buffer(Data->host.entropy, Data->size_entropy_buf / (sizeof(uint64_t)));

	if (hipMemcpyToSymbol(HIP_SYMBOL(num_bytes_find), &num_bytes, 4, 0, hipMemcpyHostToDevice) != hipSuccess)
	{
		fprintf(stderr, "hipMemcpyToSymbol to num_bytes_find failed!");
		goto Error;
	}


	static int start_save = 0;
	for (size_t step = 0; step < number_of_addresses / (Data->wallets_in_round_gpu); step++)
	{
		tools::start_time();
		if (start_save < count_save_data_in_file) {
			if (Stride->start_for_save(Config.cuda_grid, Config.cuda_block) != 0) {
				printf("Error START!!\n");
				goto Error;
			}
		}
		else
		{
			if (Stride->start(Config.cuda_grid, Config.cuda_block) != 0) {
				printf("Error START!!\n");
				goto Error;
			}
		}

		tools::generateRandomUint64Buffer(Data->host.entropy, Data->size_entropy_buf / (sizeof(uint64_t)));

		if (save_thread.joinable()) save_thread.join();

		if (start_save < count_save_data_in_file) {
			if (Stride->end_for_save() != 0) {
				printf("Error END!!\n");
				goto Error;
			}
		}
		else
		{
			if (Stride->end() != 0) {
				printf("Error END!!\n");
				goto Error;
			}
		}

		if (start_save < count_save_data_in_file) {
			start_save++;
			save_thread = std::thread(&saveResult, Save);
		}

		tools::checkResult(Data->host.ret);

		float delay;
		tools::stop_time_and_calc(&delay);
		std::cout << "\rSPEED: " << std::setw(8) << std::fixed << tools::formatWithCommas((float)Data->wallets_in_round_gpu / (delay / 1000.0f)) << " MNEMONICS/SECOND AND "
			<< tools::formatWithCommas(((float)Data->wallets_in_round_gpu * NUM_ALL_CHILDS) / (delay / 1000.0f)) << " ADDRESSES/SECOND, ROUND: " << step;
	}

	std::cout << "\n\nEND!" << std::endl;
	if (save_thread.joinable()) save_thread.join();

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return -1;
	}

	return 0;
Error:
	std::cout << "\n\nERROR!" << std::endl;
	if (save_thread.joinable()) save_thread.join();

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return -1;
	}

	return -1;
}







