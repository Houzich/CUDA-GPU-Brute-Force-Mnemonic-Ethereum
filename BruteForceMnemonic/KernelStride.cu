﻿/**
  ******************************************************************************
  * @author		Anton Houzich
  * @version	V2.0.0
  * @date		29-April-2023
  * @mail		houzich_anton@mail.ru
  * discussion  https://t.me/BRUTE_FORCE_CRYPTO_WALLET
  ******************************************************************************
  */
#include <stdafx.h>
#include <stdio.h>
#include <stdint.h>


#include "KernelStride.hpp"
#include "Helper.h"
#include <GPU.h>
#include "../Tools/utils.h"


int KernelStrideClass::bruteforce_mnemonic(uint64_t grid, uint64_t block) {
	gl_bruteforce_mnemonic << <(uint32_t)grid, (uint32_t)block, 0, dt->stream1 >> > (dt->dev.entropy, dt->dev.dev_tables, dt->dev.ret);
	return 0;
}

int KernelStrideClass::bruteforce_mnemonic_for_save(uint64_t grid, uint64_t block) {
	gl_bruteforce_mnemonic_for_save << <(uint32_t)grid, (uint32_t)block, 0, dt->stream1 >> > (dt->dev.entropy, dt->dev.dev_tables, dt->dev.ret, dt->dev.hash160, dt->dev.save);
	return 0;
}

int KernelStrideClass::memsetGlobal()
{
	if (hipMemcpyAsync(dt->dev.entropy, dt->host.entropy, dt->size_entropy_buf, hipMemcpyHostToDevice, dt->stream1) != hipSuccess) { fprintf(stderr, "hipMemcpyAsync to Board->dev.entropy failed!"); return -1; }
	if (hipMemsetAsync(dt->dev.ret, 0, sizeof(retStruct), dt->stream1) != hipSuccess) { fprintf(stderr, "hipMemset Board->dev.ret failed!"); return -1; }
	return 0;
}


int KernelStrideClass::cudaMallocDevice(uint8_t** point, uint64_t size, uint64_t* all_gpu_memory_size, std::string buff_name) {
	if (hipMalloc(point, size) != hipSuccess) {
		fprintf(stderr, "hipMalloc (%s) failed! Size: %s", buff_name.c_str(), tools::formatWithCommas(size).data()); return -1;
	}
	*all_gpu_memory_size += size;
	//if(size == 0)
	//	std::cout << "!!! WORNING !!! MALLOC GPU MEMORY SIZE (" << buff_name << "): 0.000000 MB\n";
	//else
	//	std::cout << "MALLOC GPU MEMORY SIZE (" << buff_name << "): " << std::to_string((float)size / (1024.0f * 1024.0f)) << " MB\r";
	return 0;
}

int KernelStrideClass::init()
{
	size_t memory_size = 0;
	for (int i = 0; i < 256; i++)
	{
		std::string name = "Table " + tools::byteToHexString(i);
		if (cudaMallocDevice((uint8_t**)&dt->dev.tables[i].table, dt->host.tables[i].size, &memory_size, name.c_str()) != 0)
		{
			std::cout << "Error cudaMallocDevice(), Board->dev.table[i]! i = " << i << std::endl;
			return -1;
		}
		dt->dev.tables[i].size = dt->host.tables[i].size;
		dt->dev.memory_size += dt->host.tables[i].size;
	}
	//std::cout << "MALLOC MEMORY SIZE (TABLES GPU): " << std::to_string((float)memory_size / (1024.0f * 1024.0f)) << " MB\n";

	std::cout << "INIT GPU ... \n";
	for (int i = 0; i < 256; i++)
	{
		if (hipMemcpy((void*)dt->dev.tables[i].table, dt->host.tables[i].table, dt->host.tables[i].size, hipMemcpyHostToDevice) != hipSuccess)
		{
			std::cout << "hipMemcpy to Board->dev.table[i] failed! i = " << i << std::endl;
			return -1;
		}
		const size_t percentDone = (i * 100 / 256);
		std::cout << "  " << percentDone << "%\r";
	}
	if (hipMemcpy(dt->dev.dev_tables, dt->dev.tables, 256 * sizeof(tableStruct), hipMemcpyHostToDevice) != hipSuccess) { fprintf(stderr, "hipMemcpyAsync to Board->dev.table failed!"); return -1; }
	if (deviceSynchronize("init") != hipSuccess) return -1;
	return 0;
}


int KernelStrideClass::start(uint64_t grid, uint64_t block)
{
	if (memsetGlobal() != 0) return -1;
	if (bruteforce_mnemonic(grid, block) != 0) return -1;

	return 0;
}

int KernelStrideClass::end()
{
	hipError_t cudaStatus = hipSuccess;
	if (deviceSynchronize("end") != hipSuccess) return -1; //
	cudaStatus = hipMemcpy(dt->host.ret, dt->dev.ret, sizeof(retStruct), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy ret failed!");
		return -1;
	}

	return 0;
}

int KernelStrideClass::start_for_save(uint64_t grid, uint64_t block)
{
	if (memsetGlobal() != 0) return -1;
	if (bruteforce_mnemonic_for_save(grid, block) != 0) return -1;

	return 0;
}

int KernelStrideClass::end_for_save()
{
	hipError_t cudaStatus = hipSuccess;


	if (deviceSynchronize("end_for_save") != hipSuccess) return -1; //
	cudaStatus = hipMemcpy(dt->host.save, dt->dev.save, dt->size_save_buf, hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy save failed!");
		return -1;
	}
	cudaStatus = hipMemcpy(dt->host.ret, dt->dev.ret, sizeof(retStruct), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy ret failed!");
		return -1;
	}

	return 0;
}